#include "hip/hip_runtime.h"
%%writefile vector_add.cu

#include <iostream>
#include <math.h>

// Macro for checking CUDA errors
#define CUDA_CHECK(call) \
    { \
        const hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error)); \
            exit(1); \
        } \
    }

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 10;
    float A[N], B[N], C[N];

    // Initialize vectors A and B
    for (int i = 0; i < N; ++i) {
        A[i] = i * 1.0f;
        B[i] = i * 2.0f;
    }

    float *d_a, *d_b,*d_c;
    CUDA_CHECK(hipMalloc(&d_a,N*sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_b,N*sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_c,N*sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_a,A,N*sizeof(float),hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b,B,N*sizeof(float),hipMemcpyHostToDevice));

    // --- Debugging: Print device memory before kernel launch ---
    float h_a_debug[N], h_b_debug[N], h_c_debug_before[N];
    CUDA_CHECK(hipMemcpy(h_a_debug, d_a, N*sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_b_debug, d_b, N*sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_c_debug_before, d_c, N*sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "Device A before kernel: ";
    for(int i=0; i<N; ++i) std::cout << h_a_debug[i] << " ";
    std::cout << std::endl;

    std::cout << "Device B before kernel: ";
    for(int i=0; i<N; ++i) std::cout << h_b_debug[i] << " ";
    std::cout << std::endl;

    std::cout << "Device C before kernel: ";
    for(int i=0; i<N; ++i) std::cout << h_c_debug_before[i] << " ";
    std::cout << std::endl;
    // --- End Debugging ---


    int blocksize=256;
    int gridsize=(int)ceil((float)N/blocksize); // Cast to float for ceil function
    vectorAdd<<<gridsize,blocksize>>>(d_a,d_b,d_c,N);
    CUDA_CHECK(hipGetLastError()); // Check for launch errors
    CUDA_CHECK(hipDeviceSynchronize()); // Add synchronization here

    CUDA_CHECK(hipMemcpy(C,d_c,N*sizeof(float),hipMemcpyDeviceToHost));

    // Print the final result
    std::cout << "Final result of vector addition:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << C[i] << " ";
    }
    std::cout << std::endl;

    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));

    return 0; // Add return statement to main
}
